#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <pthread.h>
#include <stdint.h>
#include <inttypes.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define SIZE 8192
#define V100CORESFP64 2560
#define WARPSIZE 32

void initOutput(double *input) {
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            input[i * SIZE + j] = 0;
        }
    }
}

void initInput(double *input1, double *input2) {
    srand(time(NULL));
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            input1[i * SIZE + j] = (double) rand();
            input2[i * SIZE + j] = (double) rand();
        }
    }
}

void printM(double *matrix) {
    for (int j = 0; j < SIZE; j++) {             //jki
        for (int i = 0; i < SIZE; i++) {
            printf("%f ", round(matrix[j * SIZE + i]));
        }
        printf("\n");
    }
}

void matmuljki(const double *input1, const double *input2, double *output) {
    for (int j = 0; j < SIZE; j++) {             //jki
        for (int k = 0; k < SIZE; k++) {
            for (int i = 0; i < SIZE; i++) {
                output[j * SIZE + i] += input1[j * SIZE + k] * input2[k * SIZE + i];
            }
        }
    }

}

int verify(double *matrix, double *test) {
    for (int j = 0; j < SIZE; j++) {             //jki
        for (int i = 0; i < SIZE; i++) {
            if (matrix[j * SIZE + i] != test[j * SIZE + i])return 0;
        }
    }
    return 1;
}
/*
void getteam() {
    int team = omp_get_team_num();
    printf("Team: %d \n,", team);
}

void getthread() {
    int thread = omp_get_thread_num();
    printf("Thread: %d \n,", thread);
}
*/

__global__ void gpu_test(){
    if(!threadIdx.x)printf("GPU is on!\n");
}

__global__ void cuda_matmulkji(double *a, double *b, double *c){
    long k = blockIdx.x;
    long j = blockIdx.y;

    if(k < SIZE && j < SIZE){
        for(long i = threadIdx.x;i < SIZE;i += WARPSIZE){
            c[j * SIZE + i] += a[j * SIZE + k] * b[k * SIZE + i];
        }
    }
}

int main(int argc, char *argv[]) {

    gpu_test<<<1,1>>>();

    double *a, *b, *c, *test;
    hipMallocManaged(&a,SIZE*SIZE*sizeof(double));
    hipMallocManaged(&b,SIZE*SIZE*sizeof(double));
    hipMallocManaged(&c,SIZE*SIZE*sizeof(double));

    test = (double *) malloc(SIZE * SIZE * sizeof(double));
    initInput(a, b);
    initOutput(test);

    matmuljki(a, b, test);

    initOutput(c);
    printf("\n");

    //a warp/thread block has 32 threads and an NVIDA V100 has 2560 FP 64 cores https://images.nvidia.com/content/volta-architecture/pdf/volta-architecture-whitepaper.pdf
    //this means we have 80 thread blocks so "micro processors" with multiple cores each
    //those 80 thread blocks match memory alignment with SIZE/80 for example 1024/80 = 6
    //the process will take exactly 6 gpu context switches/iterations to finish
    //use all 80 available thread blocks and the full warp size of 32
    int deviceId;
    hipGetDevice(&deviceId);
    hipMemPrefetchAsync(a, SIZE*SIZE*sizeof(double),deviceId);
    hipMemPrefetchAsync(b, SIZE*SIZE*sizeof(double),deviceId);
    hipMemPrefetchAsync(c, SIZE*SIZE*sizeof(double),deviceId);
    cuda_matmulkji<<<WARPSIZE*(V100CORESFP64/WARPSIZE),SIZE*SIZE>>>(a,b,c);
    hipMemPrefetchAsync(c, SIZE*SIZE*sizeof(double),hipCpuDeviceId);

    hipDeviceSynchronize();

    if (verify(c, test)) {
        printf("Verification success");
    }
    else { printf("Verification Failed check thread and team Sizes"); };

    hipFree(a);
    hipFree(b);
    hipFree(c);
    free(test);

    return 0;
}
