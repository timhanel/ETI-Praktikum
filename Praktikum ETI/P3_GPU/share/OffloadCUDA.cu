#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <pthread.h>
#include <stdint.h>
#include <inttypes.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#ifndef SIZE
#define SIZE 3840
#endif
#define V100CORES 2560
#define SMs 80
#define WARPSIZE 32
#define M 32
#define N 1

void initOutput(double *input) {
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            input[i * SIZE + j] = 0;
        }
    }
}

void initInput(double *input1, double *input2) {
    srand(time(NULL));
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            input1[i * SIZE + j] = (double) rand();
            input2[i * SIZE + j] = (double) rand();
        }
    }
}

void printM(double *matrix) {
    printf("\n");
    for (int j = 0; j < SIZE; j++) {             //jki
        for (int i = 0; i < SIZE; i++) {
            printf("%f ", round(matrix[j * SIZE + i]));
        }
        printf("\n");
    }
}

void matmuljki(const double *input1, const double *input2, double *output) {
    for (int j = 0; j < SIZE; j++) {             //jki
        for (int k = 0; k < SIZE; k++) {
            for (int i = 0; i < SIZE; i++) {
                output[j * SIZE + i] += input1[j * SIZE + k] * input2[k * SIZE + i];
            }
        }
    }

}

int verify(double *matrix, double *test) {
    for (int j = 0; j < SIZE; j++) {             //jki
        for (int i = 0; i < SIZE; i++) {
            if (abs(matrix[j * SIZE + i] / test[j * SIZE + i])<double(1-(double)1/100000000))return 0;
        }
    }
    return 1;
}

__global__ void gpu_test(){
    if(!threadIdx.x)printf("GPU is on!\n");
}

__global__ void cuda_matmulkji(double *a, double *b, double *c){
    int i= threadIdx.x;
    int iStep= blockDim.x;/*
    int jinit= blockIdx.x;
    int jStep= gridDim.x;*/
    //printf("i: %d, i Step: %d\n",i,iStep);
    //printf("j: %d, j Step: %d\n",jinit,jStep);
    for(;i<SIZE;i+=iStep){
        double tmp=0;
        for(int k=0; k < SIZE; k ++){
            tmp += a[blockIdx.x * SIZE + k] * b[k * SIZE + i];
        }
        c[blockIdx.x*SIZE + i]=tmp;
    }

            //printf("i: %d , j: %d , nr: %f \n",i,j,c[j*SIZE + i]);
}
int main(int argc, char *argv[]) {

    //gpu_test<<<1,1>>>();

    double *a, *b, *c;
    hipMallocManaged(&a,SIZE*SIZE*sizeof(double));
    hipMallocManaged(&b,SIZE*SIZE*sizeof(double));
    hipMallocManaged(&c,SIZE*SIZE*sizeof(double));

    //test = (double *) malloc(SIZE * SIZE * sizeof(double));
    initInput(a, b);
    //initOutput(test);

    //matmuljki(a, b, test);

    initOutput(c);
    printf("\n");

    //a warp/thread block has 32 threads and an NVIDA V100 has 5120 cores
    //this means we have 160 thread blocks so "micro processors" with multiple cores each
    //those 80 thread blocks match memory alignment with SIZE/160 for example 1024/160 = 6
    //the process will take exactly 6 gpu context switches/iterations to finish

    //use all 80 available thread blocks and the full warp size of 32
    int deviceId;
    hipGetDevice(&deviceId);
    hipMemPrefetchAsync(a, SIZE*SIZE*sizeof(double),deviceId);
    hipMemPrefetchAsync(b, SIZE*SIZE*sizeof(double),deviceId);
    hipMemPrefetchAsync(c, SIZE*SIZE*sizeof(double),deviceId);
    cuda_matmulkji<<<SIZE,32>>>(a,b,c);
    hipDeviceSynchronize();

    hipMemPrefetchAsync(c, SIZE*SIZE*sizeof(double),hipCpuDeviceId);

    hipDeviceSynchronize();

    /*if (verify(c, test)) {
        printf("Verification success");
    }
    else { printf("Verification Failed check thread and team Sizes"); };*/
    //printM(c);
    //printM(test);
    hipFree(a);
    hipFree(b);
    hipFree(c);
   // free(test);

    return 0;
}
